/*
 * Alexandre Maros - 2016
 *
 * Cuda Matrix Multiplication with Global Memory.
 *
 * nvcc cuda_matrix_global.cu -o cg.o
 *
 * Implemented by Alexandre Maros for learning purposes.
 * A version of this code using Shared Memory is in here:
 * https://github.com/alepmaros/cuda_matrix_multiplication
 *
 * Distributed under the MIT Lincese.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

//32x32
#define NTHREADS_X 16
#define NTHREADS_Y 32
#define THREADS_PER_BLOCK NTHREADS_X * NTHREADS_Y

/* A macro used for error checking in CUDA function calls
 * Credit to: http://stackoverflow.com/a/14038590 for the gpuErrchk macro.
 */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void matrix_mul(int *a, int *b, int *c, int a_ncolumns, int c_nlines, int c_ncolumns)
{

    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int line =  blockIdx.y * blockDim.y + threadIdx.y;

    if (column  >= c_ncolumns || line >= c_nlines)
        return;

    int i, sum = 0;


    int beginA = a_ncolumns * line;
    int beginB = column;

    for (i = 0; i < a_ncolumns; i++)
    {
        sum += a[beginA + i] * b[i * c_ncolumns + beginB];
    }

    c[line * c_ncolumns + column] = sum;
}

int main(){

int d[9] = {363,605,847,507,845,1183,675,1125,1575};
int i, j;
int h1 = 512;
int l;
//int h2 = 256;
//int h3 = 128;
for(l=0;l<9;l++){
  printf("Now we are in value %d\n", l);
    int *x, *m1, *a;
    int *d_x, *d_m1, *d_a;
    int x_nlines, x_ncolumns;
    int m1_nlines, m1_ncolumns;
    int a_nlines, a_ncolumns;

    size_t x_size, m1_size, a_size;

    hipEvent_t start, stop;
    gpuErrchk( hipEventCreate(&start) );
    gpuErrchk( hipEventCreate(&stop) );



    x_nlines = 1;
    x_ncolumns = d[l];

    m1_nlines = d[l];
    m1_ncolumns = h1;

    a_nlines = x_nlines;
    a_ncolumns = m1_ncolumns;


//    printf("a_nlines: %d\na_ncolumns: %d\nb_nlines: %d\nb_ncolumns: %d\nc_nlines: %d\nc_ncolumns: %d\n", a_nlines, a_ncolumns, b_nlines, b_ncolumns, c_nlines, c_ncolumns);
//#endif

   /* if ( a_ncolumns != b_nlines )
    {
        printf("Number of columns in Matrix A should be equals to number of lines in Matrix B\n");
        return EXIT_FAILURE;
    }*/

    x_size = x_nlines * x_ncolumns * sizeof(int);
    m1_size = m1_nlines * m1_ncolumns * sizeof(int);
    a_size = a_nlines * a_ncolumns * sizeof(int);
    gpuErrchk( hipMalloc((void **) &d_x, x_size) );
    gpuErrchk( hipMalloc((void **) &d_m1, m1_size) );
    gpuErrchk( hipMalloc((void **) &d_a, a_size) );

    x = (int *)malloc(x_size);
    m1 = (int *)malloc(m1_size);
    a = (int *)malloc(a_size);

    srand(time(0));
    memset(a, 0, a_nlines*a_ncolumns*sizeof(int));
//    printf("Enter values for A\n");
    for (i = 0; i < x_nlines; i++)
    {
        for (j = 0; j < x_ncolumns; j++)
        {
            x[i * x_ncolumns + j]=rand() % 2;
//	    printf("%d ",a[i* x_ncolumns + j]);
        }
//        printf("\n");
    }

//    printf("Enter values for B\n");
    for (i = 0; i < m1_nlines; i++)
    {
        for (j = 0; j < m1_ncolumns; j++)
        {
            m1[i * m1_ncolumns + j]=rand()%2;
//            printf("%d ",b[i* m1_ncolumns + j]);
        }
//	printf("\n");
    }
//    printf("\n");

    gpuErrchk( hipMemcpy(d_x, x, x_size, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_m1, m1, m1_size, hipMemcpyHostToDevice) );

    dim3 tbloco = dim3(
                    (int) std::ceil( (double) a_ncolumns / NTHREADS_X ),
                    (int) std::ceil ( (double) a_nlines / NTHREADS_Y ),
                    1
                );

    dim3 tthreads = dim3(
                        NTHREADS_X,
                        NTHREADS_Y,
                        1
                    );

#ifdef __DEBUG
    printf("tbloco.x: %d tbloco.y: %d tbloco.z: %d\n", tbloco.x, tbloco.y, tbloco.z);
    printf("tthreads.x: %d tthreads.y: %d\n", tthreads.x, tthreads.y);
#endif

    hipEventRecord(start);

    // kernel call
    matrix_mul<<<tbloco,tthreads>>>(d_x, d_m1, d_a, x_ncolumns, a_nlines, a_ncolumns);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipEventRecord(stop) );
    gpuErrchk( hipMemcpy(a, d_a, a_size, hipMemcpyDeviceToHost) );
    gpuErrchk( hipEventSynchronize(stop) );

    // print Matrix
    for (i = 0; i < a_nlines; i++)
    {
        for (j = 0; j < a_ncolumns; j++)
        {
            printf("%d ", a[i * a_ncolumns + j]);
        }
        printf("\n");
    }
    printf("\n");



        float milliseconds = 0;
        gpuErrchk( hipEventElapsedTime(&milliseconds, start, stop) );
        printf("The total time taken in milliseconds is :%.5f\n", milliseconds);


    free(x); free(m1); free(a);

    gpuErrchk( hipFree(d_x) );
    gpuErrchk( hipFree(d_m1) );
    gpuErrchk( hipFree(d_a) );

}

return 0;
}
